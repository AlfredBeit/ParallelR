#include "hip/hip_runtime.h"
/*   Copyright: GTC2016, Unblock Deep Neural Network Performance Limiter by CUDA.
 *   Author: Peng Zhao, Nvidia, ParallelR.com
 */

// CUDA headfile
#include "hip/hip_runtime.h"
#include "hipblas.h"
// Basic C
#include <stdlib.h>
#include <stdio.h>
// R library
#include <R.h>
#include <Rinternals.h>
#include <Rmath.h>


#ifndef DEBUG
#define DEBUG 0
#endif


// treat it as C code
extern "C" {
    SEXP gemm_cuda(SEXP A, SEXP B, SEXP transA, SEXP transB, SEXP devID);
    SEXP pmax_cuda(SEXP A, SEXP threshold, SEXP devID);
}


// CUDA: simple implementation of pmax 
__global__ void pmax_kernel(double *A, const int M, const int N, const double threshold){
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if(tid<M*N){
      A[tid] = (A[tid] > threshold)?A[tid]:0;
   }
   return;
}


// GEMM cuda call by .CAll format and simplified for DNN
SEXP gemm_cuda(SEXP A, SEXP B, SEXP transA, SEXP transB, SEXP devID)
{


   double alpha = 1.0;
   double beta  = 0.0;

   // GPU memory allocation
   double *A_host = NULL, *B_host = NULL;
   double *A_d    = NULL, *B_d    = NULL, *Rval_d = NULL;
   // matrix dimension
   int     m, n, k;
   int     mt, nt, kt, lda, ldb;
   // transform
   int     tA, tB, gpuID;
  
   // R to C 
   A_host = REAL(A);
   B_host = REAL(B);
   SEXP RdimA = getAttrib(A, R_DimSymbol);
   SEXP RdimB = getAttrib(B, R_DimSymbol);
   // original shape
   m   = INTEGER(RdimA)[0];
   k   = INTEGER(RdimA)[1];
   n   = INTEGER(RdimB)[1];
   // transposed shape
   mt  = m;
   nt  = n;
   kt  = k;
   lda = INTEGER(RdimA)[0];
   ldb = INTEGER(RdimB)[0];

   tA  = INTEGER(transA)[0];
   tB  = INTEGER(transB)[0];

   gpuID = INTEGER(devID)[0];

   // Note that cublas follows fortran order.
   hipblasOperation_t cuTransA = HIPBLAS_OP_N;
   hipblasOperation_t cuTransB = HIPBLAS_OP_N;

   if(tA == 1) {
       cuTransA = HIPBLAS_OP_T;
       mt = k;
       kt = m;
   }

   if(tB == 1) {
       cuTransB = HIPBLAS_OP_T;
       nt = INTEGER(RdimB)[0];
       kt = INTEGER(RdimB)[1];
   }

   // set GPU ID
   hipSetDevice(gpuID);
   hipblasHandle_t handle;
   hipblasCreate(&handle);

   SEXP Rval;
   PROTECT(Rval = allocVector(REALSXP, mt*nt));

   // Memory allocation in GPU
   hipMalloc(&A_d,  mt*kt*sizeof(double));
   if(NULL == A_d) {
      printf("\nNo RAM space in GPU!\n");
      goto FREE_RESOURCE;
   }
   
   hipMalloc(&B_d,  kt*nt*sizeof(double));
   if(NULL == B_d) {
      printf("\nNo RAM space in GPU!\n");
      goto FREE_RESOURCE;
   }

   hipMalloc(&Rval_d,  mt*nt*sizeof(double));
   if(NULL == Rval_d) {
      printf("\nNo RAM space in GPU!\n");
      goto FREE_RESOURCE;
   }

   // memory copy
   hipMemcpy(A_d, A_host, mt*kt*sizeof(double), hipMemcpyHostToDevice); 
   hipMemcpy(B_d, B_host, kt*nt*sizeof(double), hipMemcpyHostToDevice); 

   // cuBLAS: double precision matrix multiplication, DGEMM
   hipblasDgemm(handle, cuTransA, cuTransB, mt, nt, kt, &alpha, A_d, lda, B_d, ldb, &beta, Rval_d, mt);
   hipMemcpy(REAL(Rval), Rval_d, mt*nt*sizeof(double), hipMemcpyDeviceToHost);
   hipDeviceSynchronize();

FREE_RESOURCE:  
   hipblasDestroy(handle);
   if(A_d) {hipFree(A_d); A_d=NULL;}
   if(B_d) {hipFree(B_d); B_d=NULL;}
   if(Rval_d) {hipFree(Rval_d); Rval_d=NULL;}

   UNPROTECT(1);
   return Rval;

}


// Specified for DNN by .CAll format
SEXP pmax_cuda(SEXP A, SEXP threshold, SEXP devID)
{
   // data structure for GPU
   double *A_host = NULL;
   double *A_d = NULL;
   double gw = 0;
   int    mm = 0, nn = 0;
   int    gpuID = 0;
  
   // data transfer from R to C by pointers
   A_host = REAL(A);
   SEXP Rdim = getAttrib(A, R_DimSymbol);
   mm   = INTEGER(Rdim)[0];
   nn   = INTEGER(Rdim)[1];
   gw   = REAL(threshold)[0];
   gpuID = INTEGER(devID)[0];

   // for multiple GPU case 
   hipSetDevice(gpuID);
   
   // return value, allocated in C and can be used in R directly
   SEXP Rval;
   PROTECT(Rval = allocVector(REALSXP, mm*nn));

   // GPU memory allocation
   hipMalloc(&A_d,  mm*nn*sizeof(double));
   if(NULL == A_d) {
      printf("\nNo RAM space in GPU!\n");
      UNPROTECT(1);
      return R_NilValue;
   }
   
   // memory copy from CPU to GPU
   hipMemcpy(A_d, A_host, mm*nn*sizeof(double), hipMemcpyHostToDevice); 
   
   // CUDA: pmax, really computation parts
   pmax_kernel<<<(mm*nn-1)/512+1, 512>>>(A_d, mm, nn, gw);
   hipMemcpy(REAL(Rval), A_d, mm*nn*sizeof(double), hipMemcpyDeviceToHost); 
   hipDeviceSynchronize();

   // Free unused memory of GPU
   if(A_d) {hipFree(A_d); A_d=NULL;}

   UNPROTECT(1);
   return Rval;
}
